/**
 * \file:        gpu_funcs.cu
 * \brief:       Some kernels for cylindrical radiator finite differences
 * \author:      Hugh Delaney
 * \version:     
 * \date:        2021-04-29
 */

#include <hip/hip_runtime.h>
#include <iostream> 
#include <iomanip> 
#include <stdlib.h> 
#include <unistd.h> 
#include <sys/time.h> 

#define SHARED_DIM 12288

// KERNELS

/* --------------------------------------------------------------------------*/
/**
 * \brief:       A function that exploits the sparseness of finite difference matrices
                 to perform a very fast cylindrical finite differences. 
                 
                 Uses two separate sets of computations/arrays. One for left propagation, and one
                 for right propagation.

                 Will only be called in main if we have space in shared memory for four separate arrays.

                 Thread safe.
 *
 * \param:       a_d
 * \param:       n
 * \param:       m
 * \param:       iters
 *
 * \returns      
 */
/* ----------------------------------------------------------------------------*/
__global__ void gpu_rad_sweep6(float * a_d, unsigned int n, unsigned int m, unsigned int iters) {
  
  // Each array will have this dimension
  int shared_dims = 2*(iters+2);

  extern __shared__ float a_s[];
  
  // a_start, b_start will be for right propogation and 
  // a_end b_end will be for left propogation
  float * a_start = a_s;
  float * b_start = &a_s[2*(iters+2)];
  float * a_end = &a_s[4*(iters+2)];
  float * b_end = &a_s[6*(iters+2)];

  float * tmp;

  int tidx = threadIdx.x;
  int step = blockDim.x;
  int index = tidx;

  // Initializing the arrays
  while (index < shared_dims) {
    a_start[index] = b_start[index] = a_end[index] = b_end[index] = 0.0f;
    index += step;
  }

  a_start[0] = b_start[0] = a_end[shared_dims-2] = b_end[shared_dims-2] = (blockIdx.x+1)/ (float)n;
  a_start[1] = b_start[1] = a_end[shared_dims-1] = b_end[shared_dims-1] = 0.80f * a_start[0];


  for (int i=0;i<iters;i++) {
    __syncthreads();

    // Right propagation
    index = tidx + 2;
    while (index < shared_dims-2) {
      b_start[index] =  (1.70f*a_start[index-2] + 1.40f*a_start[index-1] + a_start[index] + 0.60f*a_start[index+1] + 0.30f*a_start[index+2])/5.0f;
      if (b_start[index]==0.0f) break;
      index += step;
    }
    
    // Left propagation
    index = shared_dims - 3 - tidx;
    while (index > 1) {
      b_end[index] =  (1.70f*a_end[index-2] + 1.40f*a_end[index-1] + a_end[index] + 0.60f*a_end[index+1] + 0.30f*a_end[index+2])/5.0f;
      if (b_end[index]==0.0f) break;
      index -= step;
    }
    
    // Swapping arrays
    tmp = a_start;
    a_start = b_start;
    b_start = tmp;

    tmp = a_end;
    a_end = b_end;
    b_end = tmp;
  }

  // Writing to global memory
  int glob_index = blockIdx.x*m + tidx;
  index = tidx;
  // Writing right prop
  while (index < shared_dims) {
    a_d[glob_index] = a_start[index];
    index += step;
    glob_index += step;
  }

  // Writing left prop
  glob_index = (blockIdx.x+1)*m - shared_dims + tidx+2;
  index = tidx;
  while (index < shared_dims - 2) {
    a_d[glob_index] = a_end[index];
    index += step;
    glob_index += step;
  }
}


/* --------------------------------------------------------------------------*/
/**
 * \brief:       Suitable for matrices that are non sparse. Loads entire rows into
                 shared memory by taking chunks of rows at a time. 
                
                 NB NOT THREAD SAFE for block size greater than 128

                 (due to lack of syncthreads in inner while loops)

                 Much slower than gpu_rad_sweep6 for lower iteration counts
 *
 * \param:       a_d
 * \param:       n
 * \param:       m
 * \param:       iters
 *
 * \returns      
 */
/* ----------------------------------------------------------------------------*/
__global__ void gpu_rad_sweep5(float * a_d, unsigned int n, unsigned int m, unsigned int iters) {
  __shared__ float a_shared[SHARED_DIM];
  int tidx = threadIdx.x;
  int step = blockDim.x;

  // These values will be used so we only need one array to do our calculations, instead
  // of two
  float f0, f1;
  float g0 = -1.0, g1 = -1.0, tmp;

  int remaining;
  int glob_index, shared_index, glob_start;

  for (unsigned int i=0;i<iters;i++) {
    glob_start = m*blockIdx.x+2;;
    remaining = m-2;
    f0 = (blockIdx.x+1)/ (float)n;
    f1 = 0.80f*f0;

    glob_index = glob_start + tidx;

    //              BEGIN LOOP          //
    //  if entire row will not fit in SHARED_DIM     
    while (remaining > SHARED_DIM-2){
      // These values will be cached from previous cycle or will hold boundary conditions
      a_shared[0] = f0;
      a_shared[1] = f1;

      shared_index = tidx+2;
      glob_index = glob_start+tidx;

      // Load section of array into shared memory
      while (shared_index < SHARED_DIM) {
        a_shared[shared_index] = a_d[glob_index];
        shared_index += step;
        glob_index += step;
      }

      __syncthreads();
      shared_index = tidx+2;

      // Perform calculation from shared[2] to shared[SHARED_DIM-3]
      //for (int i=0;i<loop_iters;i++) {
      while (shared_index < SHARED_DIM-2) {
        g0 = (1.70f*a_shared[shared_index-2] + 1.40f*a_shared[shared_index-1] + a_shared[shared_index] + 0.60f*a_shared[shared_index+1] + 0.30f*a_shared[shared_index+2])/5.0f;
        if (g1 >= 0.0f) a_shared[shared_index-step] = g1;
        // Swap g0 and g1 so that the just-computed value will be stored in the next cycle
        tmp = g0;
        g0 = g1;
        g1 = tmp;
        shared_index +=step;
      }
      __syncthreads();
      // Cache 4th and 3rd last values of prev array to store in first two vals of next array
      f0 = a_shared[SHARED_DIM-4];
      f1 = a_shared[SHARED_DIM-3];


      // Store final vals in shared array
      if (shared_index-step < SHARED_DIM) a_shared[shared_index-step] = g1;
      __syncthreads();

      // Reset g0, g1 so not used on first iteration of next run
      g1 = -1.0f, g0 = -1.0f;

      shared_index = tidx+2;
      glob_index = glob_start+tidx;

      // Write shared array to global
      while (shared_index < SHARED_DIM-2) {
        a_d[glob_index] = a_shared[shared_index];
        shared_index += step;
        glob_index += step;
      }
      __syncthreads();

      // Decrement the global index so last two values of prev array are reloaded for simplicity
      remaining -= SHARED_DIM - 4;
      glob_start += SHARED_DIM - 4;
    }
    //            END LOOP             //
    // The rest of array is now smaller than shared dim
    // These values will be cached from previous cycle or will hold boundary conditions
    if (threadIdx.x == 0) {
      a_shared[0] = f0;
      a_shared[1] = f1;
    }
    __syncthreads();

    shared_index = tidx+2;
    glob_index = glob_start+tidx;

    // Load section of array into shared memory
    //while (glob_index < (blockIdx.x+1)*m) {
    while (shared_index < remaining+2){
      a_shared[shared_index] = a_d[glob_index];
      shared_index += step;
      glob_index += step;
    }

    // Setting endpoints to be a_d[0], a_d[1]
    a_shared[remaining+2] = (float)(blockIdx.x+1) / (float)n;
    a_shared[remaining+3] = 0.80f*(blockIdx.x+1) / (float)n;
    __syncthreads();

    shared_index = tidx+2;
    // Perform calculation from shared[2] to shared[SHARED_DIM-3]
    while (shared_index < remaining+2) {
      g0 = (1.70f*a_shared[shared_index-2] + 1.40f*a_shared[shared_index-1] + a_shared[shared_index] + 0.60f*a_shared[shared_index+1] + 0.30f*a_shared[shared_index+2])/5.0f;
      if (g1 >= 0.0f) a_shared[shared_index-step] = g1;
      // Swap g0 and g1 so that the just computed value will be stored in the next cycle
      tmp = g0;
      g0 = g1;
      g1 = tmp;
      shared_index +=step;
    }

    __syncthreads();
    // Store final vals in shared array
    if (g1 >= 0.0f) a_shared[shared_index-step] = g1;
    __syncthreads();
    // Reset g0, g1 so not used on first iteration of next run
    g0 = -1.0f, g1 = -1.0f;

    shared_index = tidx+2;
    glob_index = glob_start + tidx;

    // Write shared array to global
    while (glob_index < (blockIdx.x+1)*m) {
      //while (shared_index < remaining+2){
      a_d[glob_index] = a_shared[shared_index];
      shared_index += step;
      glob_index += step;
    }
    __syncthreads();
    }
}


/* --------------------------------------------------------------------------*/
/**
 * \brief:       A simple row average function using the global memory.
                 
                 A faster function would exploit sparseness as in gpu_rad_sweep6 but
                 this runs quite fast anyway.
 *
 * \param:       A_d
 * \param:       n
 * \param:       m
 * \param:       avg_d
 *
 * \returns      
 */
/* ----------------------------------------------------------------------------*/
__global__ void gpu_get_averages(float * A_d, unsigned int n, unsigned int m, float * avg_d) {
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  float sum = 0.0f;
  if (idx < n) {
    for (int i=0;i<m;i++) sum += A_d[idx*m+i];
    avg_d[idx] = sum / (float) m;
  }
}
