#include "hip/hip_runtime.h"
#include <iostream> 
#include <iomanip> 
#include <stdlib.h> 
#include <sys/time.h> 

#include "matrix.h"


int main(int argc, char * argv[]) {

  // Default values for N, M, block_size
  unsigned int N {10}, M {10};
  int block_size = 32;

  // A boolean variable will tell us whether or not we want to print time
  int print_time {0};

  // Lots of time variables to keep track of time
  struct timeval start_time, cpu_time_rowsum, cpu_time_colsum, cpu_time_reduction, gpu_time_rowsum, gpu_time_colsum, gpu_time_reduction, cpu_time_rowsum1, cpu_time_colsum1, cpu_time_reduction1, gpu_time_rowsum1, gpu_time_colsum1, gpu_time_reduction1;
 
  // Default seed
  long unsigned int seed {123456};
  
  // Get optional parameters
  parse_command_line(argc, argv, N, M, seed, start_time, print_time, block_size);
  
  // Seed RNG
  srand48(seed);
  
  // Populate matrix with values from [-10.0, 10.0]
  float * A {(float *) malloc(sizeof(float) * N * M)};
  for (unsigned int i = 0; i < N*M; i++)
    A[i] = (float) drand48()*20.0 - 10.0;  

  std::cout << std::setw(60) << std::setfill('~') << '\n';
  std::cout << "\t\t\tCPU\n";
  std::cout << std::setw(60) << std::setfill('~') << '\n' << std::setfill(' ');
  print_matrix_CPU(A, N, M);

  float * rowsum {(float *) malloc(sizeof(float) * N)};
  float * colsum {(float *) malloc(sizeof(float) * M)};

  // CPU STUFF
  
  gettimeofday(&cpu_time_rowsum, NULL);
  sum_abs_rows_CPU(A, rowsum, N, M);
  gettimeofday(&cpu_time_rowsum1, NULL);
  
  //std::cout << "Rowsums: \n";
  //print_matrix_CPU(rowsum, N, 1);
  
  gettimeofday(&cpu_time_colsum, NULL);
  sum_abs_cols_CPU(A, colsum, N, M);
  gettimeofday(&cpu_time_colsum1, NULL);

  //std::cout << "Column sums: \n";
  //print_matrix_CPU(colsum, 1, M);


  gettimeofday(&cpu_time_reduction, NULL);
  float sum_of_rowsums_CPU = vector_reduction_CPU(rowsum, N);
  float sum_of_colsums_CPU = vector_reduction_CPU(colsum, M);
  gettimeofday(&cpu_time_reduction1, NULL);
  
  std::cout << "Sum of rowsums: " << std::setprecision(20) << sum_of_rowsums_CPU << std::endl;
  std::cout << "Sum of colsums: " << sum_of_colsums_CPU << std::endl;
  
  std::cout << '\n';

  // Calculating times
  double rowsum_time_CPU = ((double) cpu_time_rowsum1.tv_sec - cpu_time_rowsum.tv_sec) + (((double)(cpu_time_rowsum1.tv_usec - cpu_time_rowsum.tv_usec))/1000000.0);
  double colsum_time_CPU = ((double) cpu_time_colsum1.tv_sec - cpu_time_colsum.tv_sec) + (((double)(cpu_time_colsum1.tv_usec - cpu_time_colsum.tv_usec))/1000000.0);
  double reduction_time_CPU = ((double) cpu_time_reduction1.tv_sec - cpu_time_reduction.tv_sec) + (((double)(cpu_time_reduction1.tv_usec - cpu_time_reduction.tv_usec))/1000000.0);


  // GPU STUFF
  std::cout << std::setw(60) << std::setfill('~') << '\n';
  std::cout << "\t\t\tGPU\n";
  std::cout << std::setw(60) << std::setfill('~') << '\n' << std::setfill(' ');

  float * A_d, * rowsum_d, * colsum_d;

  hipMalloc((void **) &A_d, sizeof(float)*N*M);
  hipMalloc((void **) &rowsum_d, sizeof(float)*N);
  hipMalloc((void **) &colsum_d, sizeof(float)*M);

  hipMemcpy(A_d, A, sizeof(float)*N*M, hipMemcpyHostToDevice);

  dim3 dimBlock(block_size);
  dim3 dimGrid((std::max(N, M)/dimBlock.x) + (!(std::max(N,M)%dimBlock.x)?0:1));

  gettimeofday(&gpu_time_rowsum, NULL);
  sum_abs_rows_GPU<<<dimGrid, dimBlock>>>(A_d, rowsum_d, N, M);
  gettimeofday(&gpu_time_rowsum1, NULL);

  gettimeofday(&gpu_time_colsum, NULL);
  sum_abs_cols_GPU<<<dimGrid, dimBlock>>>(A_d, colsum_d, N, M);
  gettimeofday(&gpu_time_colsum1, NULL);

  hipMemcpy(rowsum, rowsum_d, sizeof(float)*N, hipMemcpyDeviceToHost);
  hipMemcpy(colsum, colsum_d, sizeof(float)*M, hipMemcpyDeviceToHost);
  
  gettimeofday(&gpu_time_reduction, NULL);
  float sum_of_rowsums_GPU = vector_reduction_GPU(rowsum_d, N, dimBlock, dimGrid);
  float sum_of_colsums_GPU = vector_reduction_GPU(colsum_d, M, dimBlock, dimGrid);
  gettimeofday(&gpu_time_reduction1, NULL);
  
  //std::cout << "Rowsums: \n";
  //print_matrix_CPU(rowsum, N, 1);
  std::cout << "GPU Sum of rowsums: " << std::setprecision(20) << sum_of_rowsums_GPU << std::endl;
  
  //std::cout << "Column sums: \n";
  //print_matrix_CPU(colsum, 1, M);
  std::cout << "GPU Sum of colsums: " << std::setprecision(20) << sum_of_colsums_GPU << std::endl;
  
  // Calculating times
  double rowsum_time_GPU = ((double) gpu_time_rowsum1.tv_sec - gpu_time_rowsum.tv_sec) + (((double)(gpu_time_rowsum1.tv_usec - gpu_time_rowsum.tv_usec))/1000000.0);
  double colsum_time_GPU = ((double) gpu_time_colsum1.tv_sec - gpu_time_colsum.tv_sec) + (((double)(gpu_time_colsum1.tv_usec - gpu_time_colsum.tv_usec))/1000000.0);
  double reduction_time_GPU = ((double) gpu_time_reduction1.tv_sec - gpu_time_reduction.tv_sec) + (((double)(gpu_time_reduction1.tv_usec - gpu_time_reduction.tv_usec))/1000000.0);
  
  if (print_time == 1) {
  
	std::cout << '\n' << std::setw(60) << std::setfill('~') << '\n';
	std::cout << "\tCPU TIME\tGPU TIME\tSPEEDUP\n";
	std::cout << std::setw(60) << std::setfill('~') << '\n' << std::setfill(' ');
	std::cout << "Rowsum" << std::setw(12) << std::setprecision(10) << rowsum_time_CPU << std::setw(12) << rowsum_time_GPU << std::setw(20) << rowsum_time_CPU/rowsum_time_GPU << std::endl;
	std::cout << "Colsum"  << std::setw(12) << colsum_time_CPU << std::setw(12) << colsum_time_GPU << std::setw(20) << colsum_time_CPU/colsum_time_GPU << std::endl;
	std::cout << "Reduce" <<  std::setw(12)<< reduction_time_CPU <<std::setw(12) << reduction_time_GPU << std::setw(20) << reduction_time_CPU/reduction_time_GPU << std::endl;
  
}
  
  std::cout << '\n';
  
  return 0;
}

