#include "hip/hip_runtime.h"
#include <iostream> 
#include <iomanip> 
#include <stdlib.h> 
#include <unistd.h> 
#include <sys/time.h> 

#include "matrix.h"

unsigned int MAX_DIM = 100000;

// CPU FUNCTIONS
float vector_reduction_CPU(const float * vector, const int n) {
  float sum = 0.0f;
  for (int i = 0; i < n; ++i) {
    sum += vector[i];
  }
  return sum;
}


void sum_abs_rows_CPU(float * matrix, float * rowsum, int N, int M) {
    // The return value will be the matrix of rowsums
    for (int i = 0; i < N; ++i) {
        rowsum[i] = 0.0f;
        for (int j = 0; j < M; ++j) {
            rowsum[i] += std::abs(matrix[i*M + j]);
        }
    }
}

void sum_abs_cols_CPU(float * matrix, float * colsum, int N, int M) {
    // The return value will be the matrix of rowsums
    for (int i = 0; i < M; ++i) {
        colsum[i] = 0.0f;
        for (int j = 0; j < N; ++j) {
            colsum[i] += std::abs(matrix[j*M + i]);
        }
    }
}

// HELPER FUNCTIONS
void parse_command_line(const int argc, char ** argv, unsigned int & n, unsigned int & m, long unsigned int & seed, struct timeval & start_time, int & print_time, unsigned int & block_size) {
  int c;
  unsigned int tmp;

  // Using getopt to parse the command line with options:
  // n - dimension of n
  // m - dimension of m
  // r - seed RNG with time(NULL)
  // h - help
  while ((c = getopt(argc, argv, "n:m:b:rth")) != -1) {
    switch(c) {
      case 'n':
        tmp = std::stoi(optarg); 
        if ((tmp > 1) && (tmp < MAX_DIM)) {
          n = tmp; 
        } else {
          std::cout << "Maximum dimension exceeded, using n = " << n << std::endl;
        }
        break;

      case 'm':
        tmp = std::stoi(optarg); 
        if ((tmp > 1) && (tmp < MAX_DIM)){
          m = tmp; 
        } else {
          std::cout << "Maximum dimension exceeded, using m = " << m << std::endl;
        }
        break;

      case 'b':
	tmp = std::stoi(optarg);
	if ((tmp > 1) && (tmp < 1025)) {
		block_size = tmp;
	} else {
  	  std::cout << "Invalid block size, using default " << block_size << std::endl;
	}
	break;
        // Seed the RNG with microsecond time
      case 'r':
        gettimeofday(&start_time, NULL);
        seed = start_time.tv_usec;
        std::cout << "Seeding with value: " << seed << "\n" << std::endl;
        break;

      case 't':
        gettimeofday(&start_time, NULL);
        print_time = 1;
        break;
      
      case 'h':
        std::cout << "Usage: ./cpu_calc [-n ndim] [-m mdim] [-r (seed with time value?)] [-h (help)]" <<std::endl;
        exit(EXIT_FAILURE);

      case '?':
        std::cerr << "Unrecognized input!\n";
        exit(EXIT_FAILURE);
    }
  }
}

void print_matrix_CPU(float * A, const unsigned int N, const unsigned int M) {
	if (N > 100 || M > 100) {
		return;
	}	

	for (int i = 0; i < N; i++) {
		std::cout << " | ";
		for (int j = 0; j < M; j++) 
				std::cout << std::setw(7) << std::setprecision(2)  << A[i*M + j];
		std::cout << " |\n";
	}
	std::cout << "\n";
}

