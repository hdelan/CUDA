#include "hip/hip_runtime.h"
/**
 * \file:        cpu_funcs.cu
 * \brief:       CUDA Assignment 1:
 *               Some CPU functions for summing rows, summing columns and performing vector reductions, as well as helpers.
 * \author:      Hugh Delaney
 * \version:     
 * \date:        2021-03-25
 */

#include <iostream> 
#include <iomanip> 
#include <stdlib.h> 
#include <unistd.h> 
#include <sys/time.h> 

#include "matrix.h"

unsigned int MAX_DIM = 100000;

// CPU FUNCTIONS


/* --------------------------------------------------------------------------*/
/**
 * \brief:       A function to perform a vector reduction in serial on CPU
 *
 * \param:       vector
 * \param:       n
 *
 * \returns      Sum
 */
/* ----------------------------------------------------------------------------*/
float vector_reduction_CPU(const float * vector, const int n) {
  float sum = 0.0f;
  for (int i = 0; i < n; ++i) {
    sum += vector[i];
  }
  return sum;
}


/* --------------------------------------------------------------------------*/
/**
 * \brief:       A function to compute the rowsums in serial on CPU
 *
 * \param:       matrix         The matrix to be rowsummed
 * \param:       rowsum         The N-length returned rowsum -- "a column vector"
 * \param:       N              The number of rows
 * \param:       M              The number of columns
 */
/* ----------------------------------------------------------------------------*/
void sum_abs_rows_CPU(float * matrix, float * rowsum, int N, int M) {
    // The return value will be the matrix of rowsums
    for (int i = 0; i < N; ++i) {
        rowsum[i] = 0.0f;
        for (int j = 0; j < M; ++j) {
            rowsum[i] += std::fabs(matrix[i*M + j]);
        }
    }
}

/* --------------------------------------------------------------------------*/
/**
 * \brief:       A function to compute column sums in serial on CPU
 *
 * \param:       matrix
 * \param:       colsum        The M-length colsum to be returned
 * \param:       N
 * \param:       M
 */
/* ----------------------------------------------------------------------------*/
void sum_abs_cols_CPU(float * matrix, float * colsum, int N, int M) {
    // The return value will be the matrix of rowsums
    for (int i = 0; i < M; ++i) {
        colsum[i] = 0.0f;
        for (int j = 0; j < N; ++j) {
            colsum[i] += std::fabs(matrix[j*M + i]);
        }
    }
}

// HELPER FUNCTIONS


/* --------------------------------------------------------------------------*/
/**
 * \brief:       A function to parse the command line for optional parameters
 *
 * \param:       argc
 * \param:       argv
 * \param:       n
 * \param:       m
 * \param:       seed
 * \param:       start_time
 * \param:       print_time
 * \param:       block_size
 */
/* ----------------------------------------------------------------------------*/
void parse_command_line(const int argc, char ** argv, unsigned int & n, unsigned int & m, long unsigned int & seed, struct timeval & start_time, int & print_time, unsigned int & block_size) {
  int c;
  unsigned int tmp;

  // Using getopt to parse the command line with options:
  // n - dimension of n
  // m - dimension of m
  // b - choose block size
  // r - seed RNG with time(NULL)
  // h - help
  while ((c = getopt(argc, argv, "n:m:b:rth")) != -1) {
    switch(c) {
      case 'n':
        tmp = std::stoi(optarg); 
        if ((tmp > 1) && (tmp < MAX_DIM)) {
          n = tmp; 
        } else {
          std::cout << "Maximum dimension exceeded, using n = " << n << std::endl;
        }
        break;

      case 'm':
        tmp = std::stoi(optarg); 
        if ((tmp > 1) && (tmp < MAX_DIM)){
          m = tmp; 
        } else {
          std::cout << "Maximum dimension exceeded, using m = " << m << std::endl;
        }
        break;

      // Choose the blocksize?
      case 'b':
	tmp = std::stoi(optarg);
	if ((tmp > 1) && (tmp < 1025)) {
		block_size = tmp;
	} else {
  	  std::cout << "Invalid block size, using default " << block_size << std::endl;
	}
	break;

        // Seed the RNG with microsecond time
      case 'r':
        gettimeofday(&start_time, NULL);
        seed = start_time.tv_usec;
        std::cout << "Seeding with value: " << seed << "\n" << std::endl;
        break;

      case 't':
        gettimeofday(&start_time, NULL);
        print_time = 1;
        break;
      
      case 'h':
        std::cout << "Usage: ./cpu_calc [-n ndim] [-m mdim] [-r (seed with time value?)] [-h (help)]" <<std::endl;
        exit(EXIT_FAILURE);

      case '?':
        std::cerr << "Unrecognized input!\n";
        exit(EXIT_FAILURE);
    }
  }
}

/* --------------------------------------------------------------------------*/
/**
 * \brief:       A function to print a matrix if it is smaller than 100 x 100
 *
 * \param:       A
 * \param:       N
 * \param:       M
 */
/* ----------------------------------------------------------------------------*/
void print_matrix_CPU(float * A, const unsigned int N, const unsigned int M) {
	//if (N > 100 || M > 100) {
	//	return;
	//}	

	for (int i = 0; i < N; i++) {
		std::cout << " | ";
		for (int j = 0; j < M; j++) 
				std::cout << std::setw(7) << std::setprecision(2)  << A[i*M + j];
		std::cout << " |\n";
	}
	std::cout << "\n";
}

